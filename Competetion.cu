#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<stdio.h>

const int rN = 100;

using namespace std;

class Runner

{
    

  public:
    // Runner's Number
    int runnerNumber=0;
    //Runner's location in competetion.
     int location = 0;
    //Runner's momentary speed.
     int speed = 0;
    //Runeer's finishing place.
    int place;
    //Runner Constructor function and Runner's starting to race.
    Runner(int number);


//Function that provides to runners move. 
__host__ __device__ void run()
{
   //test[i]->location+= test[i]->speed;
   this->location+= this->speed;
  
} 
void dtspeed(int dtspeed)
{
speed = dtspeed;
}

};
Runner::Runner(int number)
{
   runnerNumber = number;
}
////////////////////////////////////////////////////////////////
// this is the actual device routine that is run per thread
__global__ void myKernel(Runner** runner)
{
int idx = threadIdx.x+blockDim.x*blockIdx.x; // figure out which thread we are


runner[idx]->run();



}


int main()
{
// allocate host data
Runner* runner[rN];

int chck=0;


// initialize host data
for(int i = 0 ;i < rN; i++)
{
   runner[i] = new Runner(i);
   // *(runner + i)= new Runner(i);
}

//Generate device array for storing host array.
Runner** cpyrunner = (Runner**)malloc(sizeof(Runner*)*rN);

for(int i=0; i<100;i++)
{
hipMalloc((void**)&cpyrunner[i],sizeof(Runner));
hipMemcpy(cpyrunner[i],&(runner[i]),sizeof(Runner),hipMemcpyHostToDevice);
}

Runner** prunner = NULL;
hipMalloc((void**)&prunner,sizeof(Runner*)*rN);
///////////////

while(chck <= 100)
{

for(int i = 0; i < 100;i++)
{   
runner[i]->dtspeed(rand()%5 +1);  
hipMemcpy(cpyrunner[i],&(runner[i]),sizeof(Runner),hipMemcpyHostToDevice);
}
hipMemcpy(prunner,cpyrunner,sizeof(Runner*)*rN,hipMemcpyHostToDevice);

//To transfer from host to device
myKernel << < 100,1 >> > (prunner);
hipDeviceSynchronize();


for(int i=0; i<100 ;i++)
{
   hipMemcpy(&(runner[i]),cpyrunner[i],sizeof(Runner),hipMemcpyDeviceToHost);
}

//To print competetion state
for(int i=0 ; i<100; i++)
{
   int line = runner[i]->location;


   if(line == 100)
   {
   chck++;
   runner[i]->place = i + 1;
   }
   else{}

   if(chck==1)
   {
       cout << "Location : " << runner[i]->location << "speed : " << runner[i]->speed << "m/s \n";
       chck++;
   }

}

}
for(int i=0; i < 100;i++)
{
cout<< "Race Number: "<< runner[i]->runnerNumber << "Fınıshed Competetion  as = #"<<runner[i]->location<<"\n";

}


return 0;
}