#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<stdio.h>
//#include "Runner.h"




const int rN = 100;

using namespace std;

class Runner

{
    

  public:
    // Runner's Number
    int runnerNumber=0;
    //Runner's location in competetion.
    int location = 0;
    //Runner's momentary speed.
    int speed = 0;
    //Runeer's finishing place.
    int place;
    //Runner Constructor function and Runner's starting to race.
    Runner(int number);


//Function that provides to runners move. 
__host__ __device__ void run(Runner** test,int i)
{
   test[i]->location+= test[i]->speed;
} 


};
Runner::Runner(int number)
{
   runnerNumber = number;
}
////////////////////////////////////////////////////////////////
// this is the actual device routine that is run per thread
__global__ void myKernel(Runner** runner)
{
int idx = threadIdx.x+blockDim.x*blockIdx.x; // figure out which thread we are

for(int i = 0; i <= rN; i++)
{

runner[idx]->run(runner,idx);
//runner[idx]->location+= runner[idx]->speed;
printf("Runner %d, speed %d",runner[idx]->runnerNumber,runner[idx]->speed);
}

}


int main()
{
// allocate host data
//Runner **runner;
//runner = new Runner*[rN];
//Runner *cpyrunner;
Runner* runner[rN];

int chck=0;

// allocate device data
//hipMalloc(cpyrunner,sizeof(Runner*));


// initialize host data
for(int i = 0 ;i < rN; i++)
{
   runner[i] = new Runner(i);
   // *(runner + i)= new Runner(i);
}

//Generate device array for storing host array.
Runner** cpyrunner = (Runner**)malloc(sizeof(Runner*)*rN);

for(int i=0; i<100;i++)
{
hipMalloc((void**)&cpyrunner[i],sizeof(Runner));
hipMemcpy(cpyrunner[i],&(runner[i]),sizeof(Runner),hipMemcpyHostToDevice);
}

Runner** prunner = NULL;
hipMalloc((void**)&prunner,sizeof(Runner*)*rN);
hipMemcpy(prunner,cpyrunner,sizeof(Runner*)*rN,hipMemcpyHostToDevice);



///////////////

while(chck <= 99)
{
int winner = 1 ;

for(int i = 0; i < 100;i++)
{   
runner[i]->speed = rand()%5 +1;
hipMemcpy(cpyrunner[i],&(runner[i]),sizeof(Runner),hipMemcpyHostToDevice);
}
hipMemcpy(prunner,cpyrunner,sizeof(Runner*)*rN,hipMemcpyHostToDevice);

//To transfer from host to device
myKernel << < 99,0 >> > (prunner);
hipDeviceSynchronize();

//To print competetion state
for(int i=0 ; i<100; i++)
{
   int line = runner[i]->location;


   if(line == 100)
   {
   chck++;
   runner[i]->place = i + 1;
   }
   else{}

   if(chck==1)
   {
       cout << "Location : " << runner[i]->location << "speed : " << runner[i]->speed << "m/s";
   }

}

for(int i=0; i < 100;i++)
{
cout<< "Race Number: "<< runner[i]->runnerNumber << "Fınıshed Competetion  as = #"<<runner[i]->place;

}

}





return 0;
}